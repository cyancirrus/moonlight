
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void add(int n, float *x, float *y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n ) y[i] = x[i] +	y[i];
}



int main(void) {
	int N = 1<<20;
	std::vector<float> x(N, 1.0f), y(N, 2.0f);
	float *d_x, *d_y;
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));

	hipMemcpy(d_x, x.data(), N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y.data(), N*sizeof(float), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add<<<numBlocks, blockSize>>>(N, d_x, d_y);

	hipMemcpy(y.data(), d_y, N*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(d_y);
	
	std::cout << "y[0] = " << y[0] << "\n";
	std::cout << "y[N-1] = " << y[N-1] << "\n";

}


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


// threadIdx.x <- index of the thread inside the block
// blockIdx.x <- index of the block in the grid
// blockDim.x <- how many threads are in a block
// gridDim.x <- how many blocks in the grid

__global__ void add(int n, float *x, float *y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n ) y[i] = x[i] +	y[i];
}

__global__ void scale(int n, float c, float *x) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n ) x[i] = x[i] * c;
}

bool scale_item(void) {
	int N = 1 << 20;
	std::vector<float> x(N, 2.0f);
	float c = 3.14f;
	float *d_x;
	hipMalloc(&d_x, N*sizeof(float));
	hipMemcpy(d_x, x.data(), N*sizeof(float), hipMemcpyHostToDevice);
	
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	
	scale<<<numBlocks, blockSize>>>(N, c, d_x);
	hipMemcpy(x.data(), d_x, N*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_x);

	std::cout << "scaler should be 2*pi = " << x[0] << "\n";
	std::cout << "scaler should be 2*pi = " << x[N-10] << "\n";
	return true;
}

// void scale_vector(int size, float c, float *x) {
// 	scale<<numBloc	

// }

int add_print(void) {
	int N = 1<<20;
	std::vector<float> x(N, 1.0f), y(N, 2.0f);
	float *d_x, *d_y;
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	hipMemcpy(d_x, x.data(), N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y.data(), N*sizeof(float), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add<<<numBlocks, blockSize>>>(N, d_x, d_y);

	hipMemcpy(y.data(), d_y, N*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_x);
	hipFree(d_y);

	std::cout << "y[0] = " << y[0] << "\n";
	std::cout << "y[n-1] = " << y[N-1] << "\n";
	return 1;
}

int main(void) {
	add_print();
	scale_item();
}
